#include <stdio.h>
#include <stdlib.h>

#include "../../include/cuda/common.cuh"

namespace viewer {

hipError_t cuda_assert(const hipError_t code,
                        const char* const file,
                        const int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "cuda_assert: %s %s %d\n", hipGetErrorString(code),
                file, line);

        hipDeviceReset();
        exit(code);
    }

    return code;
}

}  // namespace viewer
